#include "hip/hip_runtime.h"
// #define TORCH_ASSERT_NO_OPERATORS
#include <ATen/OpMathType.h>
#include <ATen/native/GridSamplerUtils.h>
#include <ATen/native/cuda/GridSampler.cuh>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
// #include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>
#include <c10/macros/Macros.h>
#include <torch/torch.h>
#include <cmath>
#include <iostream>

// Core CUDA headers
#include <hip/hip_runtime.h>
#include <torch/extension.h>
// PyTorch CUDA headers

using namespace at::cuda::detail;
using namespace at::native;
using at::native::detail::GridSamplerInterpolation;
using at::native::detail::GridSamplerPadding;

template <typename scalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(512)
__global__ void fused_grid_sampler_3d_forward_kernel(
    const index_t count,
    const scalar_t* input,
    const scalar_t* grid,
    const scalar_t* affine_3d,
    const index_t N,
    const index_t C,
    const index_t Di,
    const index_t Hi,
    const index_t Wi,
    const index_t D,
    const index_t H,
    const index_t W,
    const float grid_xmin,
    const float grid_ymin,
    const float grid_zmin,
    const float grid_xmax,
    const float grid_ymax,
    const float grid_zmax,
    const bool is_displacement,
    scalar_t* output,
    const GridSamplerInterpolation interpolation_mode,
    const GridSamplerPadding padding_mode,
    const bool align_corners,
    const bool broadcast_input,
    const bool broadcast_affine_3d,
    const bool broadcast_grid
    ) {

    using opmath_t = at::opmath_type<scalar_t>;

    CUDA_KERNEL_LOOP_TYPE(index, count, index_t) {
        const index_t w = index % W;
        const index_t h = (index / W) % H;
        const index_t d = (index / (H * W)) % D;
        const index_t n = index / (D * H * W);
        // we have 3 coordinates for each grid point, so we multiply the index by 3
        const index_t grid_offset = 3 * (w + W * (h + H * (d + (broadcast_grid ? 0 : (D * n)))));

        // this is only affine coordinate
        opmath_t ix = 0, iy = 0, iz = 0;
        opmath_t x = 0, y = 0, z = 0;
        if (!grid) {
            // if grid is not provided, then affine matrix is multiplied to input coordinate
            // displacement is ignored
            // just affine coordiante here, we load the entire affine matrix
            const scalar_t* affine_3d_ptr = affine_3d + (broadcast_affine_3d ? 0 : (12 * n));
            // get normalized coordinate
            ix = w * (grid_xmax - grid_xmin) / (W-1) + grid_xmin;
            iy = h * (grid_ymax - grid_ymin) / (H-1) + grid_ymin;
            iz = d * (grid_zmax - grid_zmin) / (D-1) + grid_zmin;
            // apply affine matrix
            x = affine_3d_ptr[0] * ix + affine_3d_ptr[1] * iy + affine_3d_ptr[2] * iz + affine_3d_ptr[3];
            y = affine_3d_ptr[4] * ix + affine_3d_ptr[5] * iy + affine_3d_ptr[6] * iz + affine_3d_ptr[7];
            z = affine_3d_ptr[8] * ix + affine_3d_ptr[9] * iy + affine_3d_ptr[10] * iz + affine_3d_ptr[11];
        }
        else {
            // grid is provided, load the grid coordinate
            // x = grid[grid_offset];
            // y = grid[grid_offset + 1];
            // z = grid[grid_offset + 2];
            // if these are warp coordinates (`is_displacement` is false), affine matrix is ignored
            // if these are displacement coordinates, get the grid coordinates x, y, z, multiply by affine matrix, then add to displacement
            if (is_displacement) {
                // get grid coordinate
                ix = w * (grid_xmax - grid_xmin) / (W-1) + grid_xmin;
                iy = h * (grid_ymax - grid_ymin) / (H-1) + grid_ymin;
                iz = d * (grid_zmax - grid_zmin) / (D-1) + grid_zmin;
                // apply affine matrix
                if(affine_3d) {
                    const scalar_t* affine_3d_ptr = affine_3d + (broadcast_affine_3d ? 0 : (12 * n));
                    x = affine_3d_ptr[0] * ix + affine_3d_ptr[1] * iy + affine_3d_ptr[2] * iz + affine_3d_ptr[3];
                    y = affine_3d_ptr[4] * ix + affine_3d_ptr[5] * iy + affine_3d_ptr[6] * iz + affine_3d_ptr[7];
                    z = affine_3d_ptr[8] * ix + affine_3d_ptr[9] * iy + affine_3d_ptr[10] * iz + affine_3d_ptr[11];
                }
                // add to displacement
                x += grid[grid_offset];
                y += grid[grid_offset + 1];
                z += grid[grid_offset + 2];
            }
            else {
                // just get warp
                x = grid[grid_offset];
                y = grid[grid_offset + 1];
                z = grid[grid_offset + 2];
            }
        }

        // get the corresponding input x, y, z co-ordinates from grid
        ix = grid_sampler_compute_source_index(x, Wi, padding_mode, align_corners);
        iy = grid_sampler_compute_source_index(y, Hi, padding_mode, align_corners);
        iz = grid_sampler_compute_source_index(z, Di, padding_mode, align_corners);

        if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
            // get corner pixel values from (x, y, z)
            // for 4d, we used north-east-south-west
            // for 5d, we add top-bottom
            index_t ix_tnw = static_cast<index_t>(::floor(ix));
            index_t iy_tnw = static_cast<index_t>(::floor(iy));
            index_t iz_tnw = static_cast<index_t>(::floor(iz));

            index_t ix_tne = ix_tnw + 1;
            index_t iy_tne = iy_tnw;
            index_t iz_tne = iz_tnw;

            index_t ix_tsw = ix_tnw;
            index_t iy_tsw = iy_tnw + 1;
            index_t iz_tsw = iz_tnw;

            index_t ix_tse = ix_tnw + 1;
            index_t iy_tse = iy_tnw + 1;
            index_t iz_tse = iz_tnw;

            index_t ix_bnw = ix_tnw;
            index_t iy_bnw = iy_tnw;
            index_t iz_bnw = iz_tnw + 1;

            index_t ix_bne = ix_tnw + 1;
            index_t iy_bne = iy_tnw;
            index_t iz_bne = iz_tnw + 1;

            index_t ix_bsw = ix_tnw;
            index_t iy_bsw = iy_tnw + 1;
            index_t iz_bsw = iz_tnw + 1;

            index_t ix_bse = ix_tnw + 1;
            index_t iy_bse = iy_tnw + 1;
            index_t iz_bse = iz_tnw + 1;

            // get surfaces to each neighbor:
            opmath_t tnw = (ix_bse - ix)    * (iy_bse - iy)    * (iz_bse - iz);
            opmath_t tne = (ix    - ix_bsw) * (iy_bsw - iy)    * (iz_bsw - iz);
            opmath_t tsw = (ix_bne - ix)    * (iy    - iy_bne) * (iz_bne - iz);
            opmath_t tse = (ix    - ix_bnw) * (iy    - iy_bnw) * (iz_bnw - iz);
            opmath_t bnw = (ix_tse - ix)    * (iy_tse - iy)    * (iz - iz_tse);
            opmath_t bne = (ix    - ix_tsw) * (iy_tsw - iy)    * (iz - iz_tsw);
            opmath_t bsw = (ix_tne - ix)    * (iy    - iy_tne) * (iz - iz_tne);
            opmath_t bse = (ix    - ix_tnw) * (iy    - iy_tnw) * (iz - iz_tnw);

            // get input and output strides
            index_t out_sC = D * H * W;
            index_t inp_sC = Di * Hi * Wi;
            // get input and output pointers
            const scalar_t* inp_ptr_NC = input + (broadcast_input ? 0 : (n * (C * inp_sC)));
            scalar_t* out_ptr_NCDHW = output + (w + W * (h + H * (d + D * C * n))); // add batch, depth, height, width offset

            for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
                //   (c, iz_tnw, iy_tnw, ix_tnw) * tnw + (c, iz_tne, iy_tne, ix_tne) * tne
                // + (c, iz_tsw, iy_tsw, ix_tsw) * tsw + (c, iz_tse, iy_tse, ix_tse) * tse
                // + (c, iz_bnw, iy_bnw, ix_bnw) * bnw + (c, iz_bne, iy_bne, ix_bne) * bne
                // + (c, iz_bsw, iy_bsw, ix_bsw) * bsw + (c, iz_bse, iy_bse, ix_bse) * bse
                opmath_t out_acc = 0;
                if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tnw + Wi * (iy_tnw + Hi * iz_tnw)] * tnw;
                }
                if (within_bounds_3d(iz_tne, iy_tne, ix_tne, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tne + Wi * (iy_tne + Hi * iz_tne)] * tne;
                }
                if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tsw + Wi * (iy_tsw + Hi * iz_tsw)] * tsw;
                }
                if (within_bounds_3d(iz_tse, iy_tse, ix_tse, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_tse + Wi * (iy_tse + Hi * iz_tse)] * tse;
                }
                if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bnw + Wi * (iy_bnw + Hi * iz_bnw)] * bnw;
                }
                if (within_bounds_3d(iz_bne, iy_bne, ix_bne, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bne + Wi * (iy_bne + Hi * iz_bne)] * bne;
                }
                if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bsw + Wi * (iy_bsw + Hi * iz_bsw)] * bsw;
                }
                if (within_bounds_3d(iz_bse, iy_bse, ix_bse, Di, Hi, Wi)) {
                    out_acc += inp_ptr_NC[ix_bse + Wi * (iy_bse + Hi * iz_bse)] * bse;
                }
                *out_ptr_NCDHW = out_acc;
            }
        } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
            index_t ix_nearest = static_cast<index_t>(std::nearbyint(ix));
            index_t iy_nearest = static_cast<index_t>(std::nearbyint(iy));
            index_t iz_nearest = static_cast<index_t>(std::nearbyint(iz));

            index_t out_sC = D * H * W;
            index_t inp_sC = Di * Hi * Wi;

            // assign nearest neighbour pixel value to output pixel
            const scalar_t* inp_ptr_NC = input + (broadcast_input ? 0 : (n * (C * inp_sC)));
            scalar_t* out_ptr_NCDHW = output + (w + W * (h + H * (d + D * C * n))); // add batch, depth, height, width offset

            for (index_t c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCDHW += out_sC) {
                if (within_bounds_3d(iz_nearest, iy_nearest, ix_nearest, Di, Hi, Wi)) {
                    *out_ptr_NCDHW = inp_ptr_NC[ix_nearest + Wi * (iy_nearest + Hi * iz_nearest)];
                } else {
                    *out_ptr_NCDHW = static_cast<scalar_t>(0);
                }
            }
        }
    }
}

// Note [Passing pointer and offset to fastAtomicAdd]
// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
// For its internal bounds checking, fastAtomicAdd needs to know where the destination address
// lies relative to the entire tensor, so we pass the base grad_input.data and full offset information,
// including batch * channel offset (NC_offset).
template <typename scalar_t, typename index_t>
C10_LAUNCH_BOUNDS_1(256)
__global__ void fused_grid_sampler_3d_backward_kernel(
        const index_t nthreads,
        TensorInfo<const scalar_t, index_t> grad_output,
        TensorInfo<const scalar_t, index_t> input,
        TensorInfo<const scalar_t, index_t> grid,
        TensorInfo<scalar_t, index_t> grad_input,    // initialized to zeros (or unused if input_requires_grad is false)
        TensorInfo<scalar_t, index_t> grad_grid,     // initialized to empty
        const GridSamplerInterpolation interpolation_mode,
        const GridSamplerPadding padding_mode,
        bool align_corners,
        const index_t grad_input_memory_span,
        const bool input_requires_grad) {

    index_t C = input.sizes[1];
    index_t inp_D = input.sizes[2];
    index_t inp_H = input.sizes[3];
    index_t inp_W = input.sizes[4];
    index_t out_D = grid.sizes[1];
    index_t out_H = grid.sizes[2];
    index_t out_W = grid.sizes[3];

    index_t inp_sN = input.strides[0];
    index_t inp_sC = input.strides[1];
    index_t inp_sD = input.strides[2];
    index_t inp_sH = input.strides[3];
    index_t inp_sW = input.strides[4];

    index_t grid_sN = grid.strides[0];
    index_t grid_sD = grid.strides[1];
    index_t grid_sH = grid.strides[2];
    index_t grid_sW = grid.strides[3];
    index_t grid_sCoor = grid.strides[4];

    index_t gOut_sN = grad_output.strides[0];
    index_t gOut_sC = grad_output.strides[1];
    index_t gOut_sD = grad_output.strides[2];
    index_t gOut_sH = grad_output.strides[3];
    index_t gOut_sW = grad_output.strides[4];
    // gInp_* (and NC_offset below) are not really needed if input_requires_grad is false.
    int64_t gInp_sN = 0;
    int64_t gInp_sC = 0;
    int64_t gInp_sD = 0;
    int64_t gInp_sH = 0;
    int64_t gInp_sW = 0;
    if (input_requires_grad) {
        gInp_sN = grad_input.strides[0];
        gInp_sC = grad_input.strides[1];
        gInp_sD = grad_input.strides[2];
        gInp_sH = grad_input.strides[3];
        gInp_sW = grad_input.strides[4];
    }
    index_t gGrid_sW = grad_grid.strides[3];

    CUDA_KERNEL_LOOP_TYPE(index, nthreads, index_t) {
        const index_t w = index % out_W;
        const index_t h = (index / out_W) % out_H;
        const index_t d = (index / (out_H * out_W)) % out_D;
        const index_t n = index / (out_D * out_H * out_W);
        const auto grid_offset = n * grid_sN + d * grid_sD + h * grid_sH + w * grid_sW;

        // get the corresponding input x, y, z co-ordinates from grid
        scalar_t ix = grid.data[grid_offset];
        scalar_t iy = grid.data[grid_offset + grid_sCoor];
        scalar_t iz = grid.data[grid_offset + 2 * grid_sCoor];

        // multipliers for gradients on ix, iy, and iz
        scalar_t gix_mult, giy_mult, giz_mult;
        ix = grid_sampler_compute_source_index_set_grad(ix, inp_W, padding_mode, align_corners, &gix_mult);
        iy = grid_sampler_compute_source_index_set_grad(iy, inp_H, padding_mode, align_corners, &giy_mult);
        iz = grid_sampler_compute_source_index_set_grad(iz, inp_D, padding_mode, align_corners, &giz_mult);

        if (interpolation_mode == GridSamplerInterpolation::Bilinear) {
            // get corner pixel values from (x, y, z)
            // for 4d, we used north-east-south-west
            // for 5d, we add top-bottom
            index_t ix_tnw = static_cast<index_t>(std::floor(ix));
            index_t iy_tnw = static_cast<index_t>(std::floor(iy));
            index_t iz_tnw = static_cast<index_t>(std::floor(iz));

            index_t ix_tne = ix_tnw + 1;
            index_t iy_tne = iy_tnw;
            index_t iz_tne = iz_tnw;

            index_t ix_tsw = ix_tnw;
            index_t iy_tsw = iy_tnw + 1;
            index_t iz_tsw = iz_tnw;

            index_t ix_tse = ix_tnw + 1;
            index_t iy_tse = iy_tnw + 1;
            index_t iz_tse = iz_tnw;

            index_t ix_bnw = ix_tnw;
            index_t iy_bnw = iy_tnw;
            index_t iz_bnw = iz_tnw + 1;

            index_t ix_bne = ix_tnw + 1;
            index_t iy_bne = iy_tnw;
            index_t iz_bne = iz_tnw + 1;

            index_t ix_bsw = ix_tnw;
            index_t iy_bsw = iy_tnw + 1;
            index_t iz_bsw = iz_tnw + 1;

            index_t ix_bse = ix_tnw + 1;
            index_t iy_bse = iy_tnw + 1;
            index_t iz_bse = iz_tnw + 1;

            // get surfaces to each neighbor:
            scalar_t tnw = (ix_bse - ix)        * (iy_bse - iy)        * (iz_bse - iz);
            scalar_t tne = (ix        - ix_bsw) * (iy_bsw - iy)        * (iz_bsw - iz);
            scalar_t tsw = (ix_bne - ix)        * (iy        - iy_bne) * (iz_bne - iz);
            scalar_t tse = (ix        - ix_bnw) * (iy        - iy_bnw) * (iz_bnw - iz);
            scalar_t bnw = (ix_tse - ix)        * (iy_tse - iy)        * (iz - iz_tse);
            scalar_t bne = (ix        - ix_tsw) * (iy_tsw - iy)        * (iz - iz_tsw);
            scalar_t bsw = (ix_tne - ix)        * (iy        - iy_tne) * (iz - iz_tne);
            scalar_t bse = (ix        - ix_tnw) * (iy        - iy_tnw) * (iz - iz_tnw);

            scalar_t gix = static_cast<scalar_t>(0), giy = static_cast<scalar_t>(0), giz = static_cast<scalar_t>(0);
            const scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
            index_t NC_offset;
            if (input_requires_grad) {
                NC_offset = n * gInp_sN;
            }
            const scalar_t *inp_ptr_NC = input.data + n * inp_sN;
            // calculate bilinear weighted pixel value and set output pixel
            for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, NC_offset += gInp_sC, inp_ptr_NC += inp_sC) {
                scalar_t gOut = *gOut_ptr_NCDHW;

                // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
                if (input_requires_grad) {
                    safe_add_3d(grad_input.data, iz_tnw, iy_tnw, ix_tnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tnw * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_tne, iy_tne, ix_tne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tne * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_tsw, iy_tsw, ix_tsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tsw * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_tse, iy_tse, ix_tse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, tse * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_bnw, iy_bnw, ix_bnw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bnw * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_bne, iy_bne, ix_bne, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bne * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_bsw, iy_bsw, ix_bsw, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bsw * gOut,
                                            NC_offset, grad_input_memory_span);
                    safe_add_3d(grad_input.data, iz_bse, iy_bse, ix_bse, gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, bse * gOut,
                                            NC_offset, grad_input_memory_span);
                }
                // calculate grad_grid
                if (within_bounds_3d(iz_tnw, iy_tnw, ix_tnw, inp_D, inp_H, inp_W)) {
                    scalar_t tnw_val = inp_ptr_NC[iz_tnw * inp_sD + iy_tnw * inp_sH + ix_tnw * inp_sW];
                    gix -= tnw_val * (iy_bse - iy)        * (iz_bse - iz)        * gOut;
                    giy -= tnw_val * (ix_bse - ix)        * (iz_bse - iz)        * gOut;
                    giz -= tnw_val * (ix_bse - ix)        * (iy_bse - iy)        * gOut;
                }
                if (within_bounds_3d(iz_tne, iy_tne, ix_tne, inp_D, inp_H, inp_W)) {
                    scalar_t tne_val = inp_ptr_NC[iz_tne * inp_sD + iy_tne * inp_sH + ix_tne * inp_sW];
                    gix += tne_val * (iy_bsw - iy)        * (iz_bsw - iz)        * gOut;
                    giy -= tne_val * (ix        - ix_bsw) * (iz_bsw - iz)        * gOut;
                    giz -= tne_val * (ix        - ix_bsw) * (iy_bsw - iy)        * gOut;
                }
                if (within_bounds_3d(iz_tsw, iy_tsw, ix_tsw, inp_D, inp_H, inp_W)) {
                    scalar_t tsw_val = inp_ptr_NC[iz_tsw * inp_sD + iy_tsw * inp_sH + ix_tsw * inp_sW];
                    gix -= tsw_val * (iy - iy_bne)        * (iz_bne - iz)        * gOut;
                    giy += tsw_val * (ix_bne - ix)        * (iz_bne - iz)        * gOut;
                    giz -= tsw_val * (ix_bne - ix)        * (iy        - iy_bne) * gOut;
                }
                if (within_bounds_3d(iz_tse, iy_tse, ix_tse, inp_D, inp_H, inp_W)) {
                    scalar_t tse_val = inp_ptr_NC[iz_tse * inp_sD + iy_tse * inp_sH + ix_tse * inp_sW];
                    gix += tse_val * (iy - iy_bnw)        * (iz_bnw - iz)        * gOut;
                    giy += tse_val * (ix        - ix_bnw) * (iz_bnw - iz)        * gOut;
                    giz -= tse_val * (ix        - ix_bnw) * (iy        - iy_bnw) * gOut;
                }
                if (within_bounds_3d(iz_bnw, iy_bnw, ix_bnw, inp_D, inp_H, inp_W)) {
                    scalar_t bnw_val = inp_ptr_NC[iz_bnw * inp_sD + iy_bnw * inp_sH + ix_bnw * inp_sW];
                    gix -= bnw_val * (iy_tse - iy)        * (iz - iz_tse)        * gOut;
                    giy -= bnw_val * (ix_tse - ix)        * (iz - iz_tse)        * gOut;
                    giz += bnw_val * (ix_tse - ix)        * (iy_tse - iy)        * gOut;
                }
                if (within_bounds_3d(iz_bne, iy_bne, ix_bne, inp_D, inp_H, inp_W)) {
                    scalar_t bne_val = inp_ptr_NC[iz_bne * inp_sD + iy_bne * inp_sH + ix_bne * inp_sW];
                    gix += bne_val * (iy_tsw - iy)        * (iz - iz_tsw)        * gOut;
                    giy -= bne_val * (ix        - ix_tsw) * (iz - iz_tsw)        * gOut;
                    giz += bne_val * (ix        - ix_tsw) * (iy_tsw - iy)        * gOut;
                }
                if (within_bounds_3d(iz_bsw, iy_bsw, ix_bsw, inp_D, inp_H, inp_W)) {
                    scalar_t bsw_val = inp_ptr_NC[iz_bsw * inp_sD + iy_bsw * inp_sH + ix_bsw * inp_sW];
                    gix -= bsw_val * (iy - iy_tne)        * (iz - iz_tne)        * gOut;
                    giy += bsw_val * (ix_tne - ix)        * (iz - iz_tne)        * gOut;
                    giz += bsw_val * (ix_tne - ix)        * (iy        - iy_tne) * gOut;
                }
                if (within_bounds_3d(iz_bse, iy_bse, ix_bse, inp_D, inp_H, inp_W)) {
                    scalar_t bse_val = inp_ptr_NC[iz_bse * inp_sD + iy_bse * inp_sH + ix_bse * inp_sW];
                    gix += bse_val * (iy - iy_tnw)        * (iz - iz_tnw)        * gOut;
                    giy += bse_val * (ix        - ix_tnw) * (iz - iz_tnw)        * gOut;
                    giz += bse_val * (ix        - ix_tnw) * (iy        - iy_tnw) * gOut;
                }
            }

            // assuming grad_grid is contiguous
            // thus we can
            //     1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
            //     2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
            scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
            gGrid_ptr_NDHW[0] = gix_mult * gix;
            gGrid_ptr_NDHW[1] = giy_mult * giy;
            gGrid_ptr_NDHW[2] = giz_mult * giz;
        } else if (interpolation_mode == GridSamplerInterpolation::Nearest) {
            if (input_requires_grad) {
                auto ix_nearest = static_cast<index_t>(std::nearbyint(ix));
                auto iy_nearest = static_cast<index_t>(std::nearbyint(iy));
                auto iz_nearest = static_cast<index_t>(std::nearbyint(iz));

                // assign nearest neighbour pixel value to output pixel
                const scalar_t *gOut_ptr_NCDHW = grad_output.data + n * gOut_sN + d * gOut_sD + h * gOut_sH + w * gOut_sW;
                index_t NC_offset = n * gInp_sN;
                for (index_t c = 0; c < C; ++c, gOut_ptr_NCDHW += gOut_sC, NC_offset += gInp_sC) {
                    // calculate and set grad_input. See Note [Passing pointer and offset to fastAtomicAdd].
                    safe_add_3d(grad_input.data, iz_nearest, iy_nearest, ix_nearest,
                                            gInp_sD, gInp_sH, gInp_sW, inp_D, inp_H, inp_W, *gOut_ptr_NCDHW,
                                            NC_offset, grad_input_memory_span);
                }
            }
            // assuming grad_grid is contiguous
            // thus we can
            //     1. use index with gGrid_sW to directly compute gGrid_ptr_NDHW
            //     2. directly assign to gGrid_ptr_NDHW[0], gGrid_ptr_NDHW[1], gGrid_ptr_NDHW[2]
            scalar_t *gGrid_ptr_NDHW = grad_grid.data + index * gGrid_sW;
            gGrid_ptr_NDHW[0] = static_cast<scalar_t>(0);
            gGrid_ptr_NDHW[1] = static_cast<scalar_t>(0);
            gGrid_ptr_NDHW[2] = static_cast<scalar_t>(0);
        }
    }
}


torch::Tensor fused_grid_sampler_3d_forward_impl(
    const torch::Tensor &input, 
    const std::optional<torch::Tensor> affine_3d,
    const std::optional<torch::Tensor> grid,
    const int64_t out_D,
    const int64_t out_H,
    const int64_t out_W,
    const float grid_xmin, 
    const float grid_ymin,
    const float grid_zmin,
    const float grid_xmax,
    const float grid_ymax,
    const float grid_zmax,
    const bool is_displacement,
    int64_t interpolation_mode, int64_t padding_mode, bool align_corners) {
    // See NOTE [ grid_sampler Native Functions ].
    // Add checks here in case this is called instead of grid_sampler.
    //   check_grid_sampler_common(input, grid);
    //   check_grid_sampler_3d(input, grid, interpolation_mode);

    int64_t D, H, W;

    TORCH_CHECK(input.dim() == 5, "input must be 5D");
    TORCH_CHECK(input.device().is_cuda(), "input must be on CUDA");
    TORCH_CHECK(input.is_contiguous(), "input must be contiguous");
    TORCH_CHECK(grid.has_value() || affine_3d.has_value(), "one of grid or affine_3d must exist");

    // see if we need to broadcast any variable
    int64_t batch_size_max = input.size(0);
    if (affine_3d.has_value()) {
        batch_size_max = std::max(batch_size_max, affine_3d.value().size(0));
    }
    if (grid.has_value()) {
        batch_size_max = std::max(batch_size_max, grid.value().size(0));
    }
    // broadcast none by default 
    bool broadcast_input = false, broadcast_affine_3d = false, broadcast_grid = false;
    if (batch_size_max > 1) {
        if (input.size(0) == 1) {
            broadcast_input = true;
        } else if (input.size(0) != batch_size_max) {
            TORCH_CHECK(false, "input batch size must match batch size of affine_3d or grid");
        }

        // broadcast affine_3d if it exists
        if (affine_3d.has_value() && affine_3d.value().size(0) == 1) {
            broadcast_affine_3d = true;
        } else if (affine_3d.has_value() && affine_3d.value().size(0) != batch_size_max) {  
            TORCH_CHECK(false, "affine_3d batch size must match batch size of input or grid");
        }
        
        // broadcast grid if it exists
        if (grid.has_value() && grid.value().size(0) == 1) {
            broadcast_grid = true;
        } else if (grid.has_value() && grid.value().size(0) != batch_size_max) {
            TORCH_CHECK(false, "grid batch size must match batch size of input or affine_3d");
        }
    }

    // D, H, W will be determined by grid
    if (grid.has_value()) {
        check_grid_sampler_common(input, grid.value());
        check_grid_sampler_3d(input, grid.value(), interpolation_mode);
        TORCH_CHECK(grid.value().is_contiguous(), "grid must be contiguous");
        D = grid.value().size(1);
        H = grid.value().size(2);
        W = grid.value().size(3);
    }
    else {
        // grid does not exist, affine must exist
        // size of output is determined by input (coordinates determined by affine)
        D = out_D;
        H = out_H;
        W = out_W;
    }

    if (affine_3d.has_value()) {
        TORCH_CHECK(input.size(0) == affine_3d.value().size(0), "input and affine_3d must have the same batch size");
        TORCH_CHECK(affine_3d.value().dim() == 3, "affine_3d must be (B, 3, 4)");
        TORCH_CHECK(affine_3d.value().device().is_cuda(), "affine_3d must be on CUDA");
        TORCH_CHECK(affine_3d.value().is_contiguous(), "affine_3d must be contiguous");
        TORCH_CHECK(affine_3d.value().size(1) == 3, "affine_3d must be (B, 3, 4)");
        TORCH_CHECK(affine_3d.value().size(2) == 4, "affine_3d must be (B, 3, 4)");
    }

    // define output
    int64_t N = batch_size_max;
    int64_t C = input.size(1);
    torch::Tensor output = torch::zeros({batch_size_max, C, D, H, W}, input.options());

    // input size parameters
    int64_t count = N * D * H * W;

    // input spatial size parameters
    int64_t Di = input.size(2);
    int64_t Hi = input.size(3);
    int64_t Wi = input.size(4);

    if (count > 0) {
        AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16,
        input.scalar_type(), "fused_grid_sampler_3d_forward_kernel", [&] {
            // check if grid is 32-bit
            bool grid32bit;
            if(grid.has_value()) {
                grid32bit = canUse32BitIndexMath(grid.value());
            } else {
                grid32bit = true;
            }
            if (canUse32BitIndexMath(input) && grid32bit &&
                canUse32BitIndexMath(output)) {
                fused_grid_sampler_3d_forward_kernel<scalar_t>
                <<<GET_BLOCKS(count, 512), 512, 0, at::cuda::getCurrentCUDAStream()>>>(
                    static_cast<int>(count),
                    input.data_ptr<scalar_t>(),
                    grid.has_value() ? grid.value().data_ptr<scalar_t>() : nullptr,
                    affine_3d.has_value() ? affine_3d.value().data_ptr<scalar_t>() : nullptr,
                    static_cast<int>(N), static_cast<int>(C), static_cast<int>(Di), static_cast<int>(Hi), static_cast<int>(Wi),
                    static_cast<int>(D), static_cast<int>(H), static_cast<int>(W),
                    grid_xmin, grid_ymin, grid_zmin, grid_xmax, grid_ymax, grid_zmax,
                    is_displacement,
                    // output
                    output.data_ptr<scalar_t>(),
                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                    static_cast<GridSamplerPadding>(padding_mode),
                    align_corners,
                    broadcast_input,
                    broadcast_affine_3d,
                    broadcast_grid
                );
                C10_CUDA_KERNEL_LAUNCH_CHECK();
            } else {
                fused_grid_sampler_3d_forward_kernel<scalar_t>
                <<<GET_BLOCKS(count, 512), 512, 0, at::cuda::getCurrentCUDAStream()>>>(
                    count,
                    input.data_ptr<scalar_t>(),
                    grid.has_value() ? grid.value().data_ptr<scalar_t>() : nullptr,
                    affine_3d.has_value() ? affine_3d.value().data_ptr<scalar_t>() : nullptr,
                    N, C, Di, Hi, Wi,
                    D, H, W,
                    grid_xmin, grid_ymin, grid_zmin, grid_xmax, grid_ymax, grid_zmax,
                    is_displacement,
                    output.data_ptr<scalar_t>(),
                    static_cast<GridSamplerInterpolation>(interpolation_mode),
                    static_cast<GridSamplerPadding>(padding_mode),
                    align_corners,
                    broadcast_input,
                    broadcast_affine_3d,
                    broadcast_grid
                    );
                C10_CUDA_KERNEL_LAUNCH_CHECK();
            }
        });
    }
    return output;
}

